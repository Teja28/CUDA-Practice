#include "hip/hip_runtime.h"
int ** ReverseMatrix(int** matrix)
{

}

//assign the number of threads as number of elements in the kernel, number of blocks
__global__ void Convolution(int** matrixA, int** matrixB, int pointC)
{
  pointC = 0;
  for (int i = threadIdx.y+blockIdx.y*blockDim.y; i < N; i+=blockIdx.y*gridDim.y)
  {
    for (int j = threadIdx.x+blockIdx.x*blockDim.x; j < N; j+=blockIdx.x*gridDim.x)
    {
      pointC = pointC + (matrixA[i][j]*matrixB[j][i]);
    }
  }
}
